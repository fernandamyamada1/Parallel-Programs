#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <sys/time.h>


typedef float vtype;

typedef struct { 
    int size; 
    vtype* elements; 
} Vector; 
 
#define BLOCK_SIZE 512
#define VECTOR_SIZE 1024*1024*10
#define N_BLOCKS VECTOR_SIZE/BLOCK_SIZE
 

__global__ void SumReductionKernel(Vector v, vtype *results) 
{ 

int j;
int idy=blockDim.y*blockIdx.y+threadIdx.y;
int range=VECTOR_SIZE/(N_BLOCKS*BLOCK_SIZE);

__shared__ float vS[2*BLOCK_SIZE];
vS[threadIdx.x]=v.elements[threadIdx.x+(blockDim.x*2)*blockIdx.x];
vS[threadIdx.x+blockDim.x]=v.elements[threadIdx.x+blockDim.x+(blockDim.x*2)*(blockIdx.x+1)];
__syncthreads(); //garantir que todas as threads recebam os seus dados, para nao somar posicoes vazias
//Reducao -multiplas iteracoes
int desloc;
for(desloc=blockDim.x;desloc>0;desloc/=2){
if(threadIdx/desloc==0){
vS[threadIdx.x]+=vS[threadIdx.x+desloc];
}
__syncthreads();




}

int main() {

  int i;

  Vector v_h;
  v_h.size = VECTOR_SIZE;
  v_h.elements = (vtype *)malloc( v_h.size * sizeof(vtype)); 
  
  for (i=0; i<v_h.size; i++) {
    v_h.elements[i] = rand()%10;
  }

  struct timeval t1;
  gettimeofday(&t1, 0);

    // Load A and B to device memory 
    Vector v_d; 
    v_d.size = v_h.size;
    hipMalloc((void**)&v_d.elements, v_d.size*sizeof(vtype)); 
    hipMemcpy(v_d.elements, v_h.elements, v_d.size*sizeof(vtype), hipMemcpyHostToDevice); 

    printf("Allocated %.3f MB of memory.\n", v_d.size*sizeof(vtype)/1024./1024.);

    int nBlocks = N_BLOCKS;
    int nThreadsPerBlock = BLOCK_SIZE;
    vtype *results_d, *results_h;
    hipMalloc((void**)&results_d, nBlocks * sizeof(vtype)); 
    results_h = (vtype *)malloc(nBlocks * sizeof(vtype));

    struct timeval t2;
    gettimeofday(&t2, 0);


    // Invoke kernel 
    SumReductionKernel<<<nBlocks, nThreadsPerBlock>>>(v_d, results_d); 

    hipDeviceSynchronize();

    struct timeval t3;
    gettimeofday(&t3, 0);
 
    // Read results from device memory 
    hipMemcpy(results_h, results_d, nBlocks * sizeof(vtype), hipMemcpyDeviceToHost); 

    // Free device memory 
    hipFree(results_d); 
    hipFree(v_d.elements); 

    double result = 0;
    for (int i=0; i<N_BLOCKS; i++) {
      result += results_h[i];
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
      printf("Error: %s\n", hipGetErrorString(error));
      exit(-1);
    } 

    struct timeval t4;
    gettimeofday(&t4, 0);


  // Evaluates in the CPU
  double result_cpu = 0;
  for (int i=0; i<v_h.size; i++)
    result_cpu += v_h.elements[i];
 
 
  struct timeval t5;
  gettimeofday(&t5, 0);

  printf("transfer=%8.2f\n", (t2.tv_sec*1000. + t2.tv_usec/1000.) - (t1.tv_sec*1000. + t1.tv_usec/1000.));
  printf("     gpu=%8.2f\n", (t3.tv_sec*1000. + t3.tv_usec/1000.) - (t2.tv_sec*1000. + t2.tv_usec/1000.));
  printf("transfer=%8.2f\n", (t4.tv_sec*1000. + t4.tv_usec/1000.) - (t3.tv_sec*1000. + t3.tv_usec/1000.));
  printf("     cpu=%8.2f\n", (t5.tv_sec*1000. + t5.tv_usec/1000.) - (t4.tv_sec*1000. + t4.tv_usec/1000.));
  printf("   total=%8.2f\n", (t5.tv_sec*1000. + t5.tv_usec/1000.) - (t1.tv_sec*1000. + t1.tv_usec/1000.));

  //Checks the result
  printf ( "sum_cpu=%.2f sum_gpu=%.2f diff=%.5f\n", result_cpu, result, abs(result - result_cpu)/result_cpu);
  assert( abs(result - result_cpu)/result_cpu < 0.01);

  free (v_h.elements);
 
  return 0;
}
