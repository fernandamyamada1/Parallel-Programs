
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cassert>
#include <sys/time.h>

struct timeval tGS, tGF, tKS, tKF, tCS, tCF;

// Matrices are stored in row-major order: 
// M(row, col) = M.elements[row * M.width + col] 
typedef struct { 
    int width; 
    int height; 
    float* elements; 
} Matrix; 


// Matrix sum kernel, called by MatrixSum() 
__global__ void MatSumKernel(Matrix A, Matrix B, Matrix C) 
{ 
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;

  C.elements[idx+C.width*idy] =      
       A.elements[idx+A.width*idy] + B.elements[idx+B.width*idy];
} 

 
// Host code
void MatSum(const Matrix A, const Matrix B, Matrix C) 
{ 

    // Allocate A, B and C in device memory
    int size = A.width*A.height * sizeof(float);
    Matrix d_A; Matrix d_B; Matrix d_C;
    d_A.width = A.width; d_A.height = A.height;
    d_B.width = B.width; d_B.height = B.height;
    d_C.width = C.width; d_C.height = C.height;
    hipMalloc((void**)&(d_A.elements), size);
    hipMalloc((void**)&(d_B.elements), size);
    hipMalloc((void**)&(d_C.elements), size);

    // Load A and B to device memory  
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);      
    hipMemcpy(d_C.elements, C.elements, size, hipMemcpyHostToDevice);            

    // Invoke kernel 
    gettimeofday(&tKS, 0);
    dim3 nThreads(16, 16, 1);
    dim3 nBlocks(d_A.width/16, d_B.height/16, 1);
    MatSumKernel<<<nBlocks, nThreads>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    gettimeofday(&tKF, 0);
 
    // Read C from device memory 
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);            
 
    // Free device memory 
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
} 
 
int main(int argc, char **argv) {

  if (argc < 3) {
    printf ("Usage: %s nLines nColumns\n", argv[0]);
    return -1;
  }

  int nLines   = atoi (argv[1]);
  int nColumns = atoi (argv[2]);  
  int i,j;

  Matrix A, B, C;
  A.width = nColumns; A.height = nLines;
  B.width = nColumns; B.height = nLines;
  C.width = nColumns; C.height = nLines;
 
  A.elements = (float *)malloc(A.width*A.height * sizeof(float)); 
  B.elements = (float *)malloc(B.width*B.height * sizeof(float)); 
  C.elements = (float *)malloc(C.width*C.height * sizeof(float)); 
  
  for (i=0; i<A.width*A.height; i++)
    A.elements[i] =   i   % 10 + 1;
  for (i=0; i<B.width*B.height; i++)
    B.elements[i] = (i+1) % 11 + 1;
  for (i=0; i<C.width*C.height; i++)
    C.elements[i] = 0;



  // Performs the sum
  gettimeofday(&tGS, 0);
  MatSum(A, B, C);
  gettimeofday(&tGF, 0);

  //Checks the result
  gettimeofday(&tCS, 0);
  for (i=0; i<nColumns; i++) {
    for (j=0; j<nLines; j++) {
      float ctmp = A.elements[j*nColumns+i] + B.elements[j*nColumns+i];
      assert( fabs(ctmp - C.elements[j*nColumns+i]) < 0.001);
    }
  }
  gettimeofday(&tCF, 0);

  printf(" total=%8.2f\n", (tCF.tv_sec*1000. + tCF.tv_usec/1000.) - (tGS.tv_sec*1000. + tGS.tv_usec/1000.));
  printf("kernel=%8.2f\n", (tKF.tv_sec*1000. + tKF.tv_usec/1000.) - (tKS.tv_sec*1000. + tKS.tv_usec/1000.));
  printf("   gpu=%8.2f\n", (tGF.tv_sec*1000. + tGF.tv_usec/1000.) - (tGS.tv_sec*1000. + tGS.tv_usec/1000.));
  printf("   cpu=%8.2f\n", (tCF.tv_sec*1000. + tCF.tv_usec/1000.) - (tCS.tv_sec*1000. + tCS.tv_usec/1000.));

  return 0;
}

