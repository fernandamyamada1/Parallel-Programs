#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <sys/time.h>

struct timeval tGS, tGF, tKS, tKF, tCS, tCF;

// Matrices are stored in row-major order: 
// M(row, col) = M.elements[row * M.width + col]
typedef struct { 
    int width; 
    int height; 
    float* elements; 
} Matrix; 
 

// Matrix multiplication kernel called by MatrixMul() 
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) 
{
int j;
int idx=blockDim.x*blockIdx.x+threadIdx.x;
int idy=blockDim.y*blockIdx.y+threadIdx.y;

for(j=0;j<A.width;j++){
C.elements[idx*C.width+idy]+=A.elements[idx*A.width+j]*B.elements[j*B.width+idy]
}

     
} 
  
// Matrix multiplication - Host code 
void MatMul(const Matrix A, const Matrix B, Matrix C) 
{ 

    // Load A and B to device memory
Matrix *d_a, *d_b, *d_c;

hipMalloc(&(d_A.elements), A.width*A.height*sizeof(float));
hipMalloc(&(d_B.elements), B.width*B.height*sizeof(float));
hipMalloc(&(d_C.elements), C.width*C.height*sizeof(float));

int sizeA= A.width*A.height*sizeof(float);
int sizeB= B.width*B.height*sizeof(float);
int sizeC= C.width*C.height*sizeof(float);

hipMemcpy(d_A.elements, A.elements, sizeA, hipMemcpyHostToDevice);
hipMemcpy(d_B.elements, B.elements, sizeB, hipMemcpyHostToDevice);
hipMemcpy(d_C.elements, C.elements, sizeC, hipMemcpyHostToDevice);
//Define num threads and block size

dim3 nThreads(1,16,1);
dim3 nBlocks(A.height/nThreads.x,C.width/nThreads.y,1);




    // Invoke kernel 
    gettimeofday(&tKS, 0);
    MatMulKernel<<<nBlocks, nThreadsPerBlock>>>(d_A, d_B, d_C);
    gettimeofday(&tKF, 0);





 
    // Read C from device memory 

hipMemcpy(C.elements, d_C.elements, sizeC, hipMemcpyDeviceToHost);


 
    // Free device memory 
hipFree((d_A.elements));
hipFree((d_B.elements));
hipFree((d_C.elements));





} 


int main(int argc, char **argv) {

 if (argc < 3) {
    printf ("Usage: %s nLines nColumns\n", argv[0]);
    return -1;
  }

  int nLines   = atoi (argv[1]);
  int nColumns = atoi (argv[2]);  
  int i,j,k;

  Matrix A, B, C;
  A.width = nColumns; A.height = nLines;
  B.width = nColumns; B.height = nLines;
  C.width = nColumns; C.height = nLines;
 
  A.elements = (float *)malloc(A.width*A.height * sizeof(float)); 
  B.elements = (float *)malloc(B.width*B.height * sizeof(float)); 
  C.elements = (float *)malloc(C.width*C.height * sizeof(float)); 
  
  for (i=0; i<A.width*A.height; i++)
    A.elements[i] =   i   % 10 + 1;
  for (i=0; i<B.width*B.height; i++)
    B.elements[i] = (i+1) % 11 + 1;

  struct timeval t1;
  gettimeofday(&t1, 0);

  // Perform the multiplication
  gettimeofday(&tGS, 0);
  MatMul(A, B, C);
  gettimeofday(&tGF, 0);

  //Checks the result
  gettimeofday(&tCS, 0);
  for (i=0; i<nLines; i++) {
    for (j=0; j<nLines; j++) {
      float ctmp = 0;
      for (k=0; k<nLines; k++)
	ctmp += A.elements[i*nLines+k] * B.elements[k*nLines+j];
      assert( fabs(ctmp - C.elements[i*nLines+j]) < 0.001);
    }
  }
  gettimeofday(&tCF, 0);

  printf(" total=%8.2f\n", (tCF.tv_sec*1000. + tCF.tv_usec/1000.) - (tGS.tv_sec*1000. + tGS.tv_usec/1000.));
  printf("kernel=%8.2f\n", (tKF.tv_sec*1000. + tKF.tv_usec/1000.) - (tKS.tv_sec*1000. + tKS.tv_usec/1000.));
  printf("   gpu=%8.2f\n", (tGF.tv_sec*1000. + tGF.tv_usec/1000.) - (tGS.tv_sec*1000. + tGS.tv_usec/1000.));
  printf("   cpu=%8.2f\n", (tCF.tv_sec*1000. + tCF.tv_usec/1000.) - (tCS.tv_sec*1000. + tCS.tv_usec/1000.));


  return 0;
}

